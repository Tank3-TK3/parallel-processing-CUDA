//////////////////////////////////////////////////
//           Program 04 Query Device            //
//////////////////////////////////////////////////
#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

char* getDeviceArchitecture( hipDeviceProp_t devProp )
{
	char* sign = "";
	switch( devProp.major )
	{
	case 2:
		sign = "Fermi";
		break;
	case 3:
		sign = "Kepler";
		break;
	case 5:
		sign = "Maxwell";
		break;
	case 6:
		sign = "Pascal";
		break;
	case 7:
		sign = "Volta or Turing";
		break;
	case 8:
		sign = "Ampere";
	default:
		sign = "Unknown device type";
		break;
	}
	return sign;
}

int getSPcores( hipDeviceProp_t devProp )
{
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch( devProp.major )
    {
        case 2:
            if( devProp.minor == 1 ) cores = mp * 48;
            else cores = mp * 32;
            break;
        case 3:
            cores = mp * 192;
            break;
        case 5:
            cores = mp * 128;
            break;
        case 6:
            if( ( devProp.minor == 1 ) || ( devProp.minor == 2 ) ) cores = mp * 128;
            else if( devProp.minor == 0 ) cores = mp * 64;
            else cout << "Unknown device type\n";
            break;
        case 7:
            if( ( devProp.minor == 0 ) || ( devProp.minor == 5 ) ) cores = mp * 64;
            else cout << "Unknown device type\n";
            break;
        case 8:
            if( devProp.minor == 0 ) cores = mp * 64;
            else if( devProp.minor == 6 ) cores = mp * 128;
            else cout << "Unknown device type\n";
            break;
        default:
            cout << "Unknown device type\n"; 
            break;
    }
    return cores;
}

void printDevProp( int i )
{
	hipDeviceProp_t devProp;
	hipGetDeviceProperties( &devProp, i );
	cout << " - ASCII string identifying device: " << devProp.name << "\n";
	cout << " - Device architecture name: " << getDeviceArchitecture( devProp ) << "\n";
	cout << " - Major compute capability: " << devProp.major << "\n";
	cout << " - Minor compute capability: " << devProp.minor << "\n";
	cout << " - Number of multiprocessors on device: " << devProp.multiProcessorCount << "\n";
	cout << " - Cores CUDA: %d\n" << getSPcores(devProp);
	cout << "Total de memoria global:           %u\n" << devProp.totalGlobalMem;
	cout << "Total de memoria compartida por bloque: %u\n" << devProp.sharedMemPerBlock;
	cout << "Total de registros por bloque:     %d\n" << devProp.regsPerBlock;
	cout << "Tamaño del warp:                     %d\n" << devProp.warpSize;
	cout << "Pitch maximo de memoria:          %u\n" << devProp.memPitch;
	cout << "Hilos maximos por bloque:     %d\n" << devProp.maxThreadsPerBlock;
	for ( int i = 0 ; i < 3 ; ++i )
		cout << "Dimension maxima %d de grid:   %d\n" << i, devProp.maxGridSize[i];
	for ( int i = 0 ; i < 3 ; ++i )
		cout << "Dimension maxima %d de bloque:  %d\n" << i, devProp.maxThreadsDim[i];
	cout << "Velocidad del reloj:                    %d\n" << devProp.clockRate;
	cout << "Memoria constante total:         %u\n" << devProp.totalConstMem;
	cout << "Alineamiento de textura:             %u\n" << devProp.textureAlignment;
	cout << "Copiado y ejecucion concurrente: %s\n" << (devProp.deviceOverlap ? "Si" : "No");
	cout << "Numero de multiprocesadores:     %d\n" << devProp.multiProcessorCount;
	cout << "Timeout de ejecucion del Kernel:      %s\n" << (devProp.kernelExecTimeoutEnabled ? "Si" : "No");
}

int main( int argc, char* argv[] )
{
	int devCount;
	hipGetDeviceCount( &devCount );

	cout << "##################################################\n";
	cout << "\t  > CUDA Device Specifications <\n";
	cout << "\t     (Total CUDA devices: " << devCount << ")\n";

	for ( int i = 0 ; i < devCount ; ++i )
	{
		cout << "##################################################\n";
		// Get device properties
		cout << "+ CUDA device: " << i << "\n";
		printDevProp( i );
		cout << "##################################################\n\n";
	}

	system( "pause" );
	return 0;
}
