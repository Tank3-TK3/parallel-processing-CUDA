//////////////////////////////////////////////////
//          Project 01 PI Calculation           //
//////////////////////////////////////////////////
// C++
#include <cstdlib>
#include <cstdio>
#include <ctime>
#include <cmath>
// CUDA C / C++

#include <hip/hip_runtime.h>


const double iterations = 90000000; /*MAX: 94906264*/

__host__ int printDevProp()
{
	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, 0);
	printf("==================================================\n");
	printf(" >>>>>>> PI Calculation with CPU and GPU <<<<<<<\n");
	printf(" - Device Name: %s\n", devProp.name);
	printf(" - Maximum number of threads per block: %d\n", devProp.maxThreadsPerBlock);
	printf(" - Number of iterations: %.1lf\n", iterations);
	printf("==================================================\n");
	return devProp.maxThreadsPerBlock;
}

__host__ double piCPU()
{
	double sum = 0;
	for (double i = 1; i < iterations; ++i)
	{
		sum += (1 / (i * i));
	}
	return sqrt(sum * 6);
}

int main(int argc, char* argv[])
{
	hipSetDevice(0);

	int maxHilos = printDevProp();

	printf("\t\t<<<<< CPU >>>>>\n");
	clock_t timer1 = clock();
	printf(" - The value of PI in CPU is: %.8lf\n", piCPU()); /*MAX DEC: 51*/
	timer1 = clock() - timer1;
	printf(" - Total CPU time: %f ms.\n", ((((double)timer1) / CLOCKS_PER_SEC) * 1000));
	printf("==================================================\n");

	system("pause");
	return 0;
}
