#include "hip/hip_runtime.h"
//////////////////////////////////////////////////
//          Project 01 PI Calculation           //
//////////////////////////////////////////////////
// C++
#include <cstdlib>
#include <cstdio>
#include <ctime>
#include <cmath>
// CUDA C / C++
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

const double iterations = 90000000; /*MAX: 94906264*/

__host__ int printDevProp()
{
	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, 0);
	printf("==================================================\n");
	printf(" >>>>>>> PI Calculation with CPU and GPU <<<<<<<\n");
	printf(" - Device Name: %s\n", devProp.name);
	printf(" - Maximum number of threads per block: %d\n", devProp.maxThreadsPerBlock);
	printf(" - Number of iterations: %.1lf\n", iterations);
	printf("==================================================\n");
	return devProp.maxThreadsPerBlock;
}

__host__ double piCPU()
{
	printf("\t\t<<<<< CPU >>>>>\n");
	double sum = 0;
	clock_t timer1 = clock();
	for (double i = 1; i < iterations; ++i)
	{
		sum += (1 / (i * i));
	}
	timer1 = clock() - timer1;
	double pi = sqrt(sum * 6);
	printf(" - The value of PI in CPU is: %.8lf\n", pi); /*MAX DEC: 51*/
	printf(" - Total CPU time: %f ms.\n", ((((double)timer1) / CLOCKS_PER_SEC) * 1000.0));
	printf("==================================================\n");
	return pi;
}

__host__ int printMenuOpt()
{
	int opt = ' ';
	printf(" - Write the number of the case you want to run:\n");
	printf("   1) Case01 - 1 Block with m Thread.\n");
	printf("   2) Case02 - x Blocks with 1 Thread each.\n");
	printf("   3) Case03 - x Blocks with m Threads each.\n");
	printf("   4) Case04 - x * y Blocks with 1 Thread each.\n");
	printf("   5) Case05 - 1 Block with m * n Threads each.\n");
	printf("   6) Case06 - x * y Blocks with m * n Threads each.\n");
	printf("> ");
	scanf("%d", &opt);
	return opt;
}

__global__ void sum01GPU01(double* arrayGPU, double iter)
{
	int tid = threadIdx.x;
	double segment = ((iter / 1024) * tid) + 1;
	for (double i = segment; i < (segment + (iter / 1024)) - 1; ++i)
	{
		arrayGPU[tid] = arrayGPU[tid] + (1 / (i * i));
	}
}

__global__ void sum02GPU01(double* arrayGPU, double* numpiGPUCPU)
{
	for (int i = 1; i < 1024; ++i)
	{
		arrayGPU[0] = arrayGPU[0] + arrayGPU[i];
	}
	numpiGPUCPU[0] = sqrt(arrayGPU[0] * 6);
}

__global__ void piGPU02()
{

}

__global__ void piGPU03()
{

}

__global__ void piGPU04()
{

}

__global__ void piGPU05()
{

}

__global__ void piGPU06()
{

}

__host__ void printStats(clock_t timer, dim3 dimGrid, dim3 dimBlock)
{
	printf(" - Total Threads: %d\n",
		dimGrid.x * dimGrid.y * dimGrid.z * dimBlock.x * dimBlock.y * dimBlock.z);
	printf(" - Configuracion de ejecucion: \n");
	printf("   + Grid [%d, %d, %d] Bloque [%d, %d, %d]\n",
		dimGrid.x, dimGrid.y, dimGrid.z, dimBlock.x, dimBlock.y, dimBlock.z);
}

int main(int argc, char* argv[])
{
	clock_t timer;
	dim3 dimGrid;
	dim3 dimBlock;
	hipError_t cudaStatus;
	double* numpiCPUGPU[1];
	double* numpiGPUCPU;

	hipFree(0);
	hipSetDevice(0);

	int maxHilos = printDevProp();

	switch (printMenuOpt())
	{
	case 1: // Case01 - 1 Block with m Thread.
		printf("==================================================\n");
		printf("\t\t<<<<< GPU >>>>>\n");
		double* arrayCPU[87891];
		double* arrayGPU;
		hipMalloc((void**)&arrayGPU, (87891) * sizeof(double));
		hipMalloc((void**)&numpiGPUCPU, 1 * sizeof(double));
		hipMemset(arrayGPU, 0, (87891) * sizeof(double));
		hipMemcpy(arrayGPU, arrayCPU, iterations * sizeof(double), hipMemcpyHostToDevice);
		dimGrid = { 1 , 1 , 1 };
		dimBlock = { 1024 , 1 , 1 };
		timer = clock();
		sum01GPU01 << <dimGrid, dimBlock >> > (arrayGPU, iterations);
		hipDeviceSynchronize();
		dimBlock = { 1 , 1 , 1 };
		hipMemcpy(numpiGPUCPU, numpiCPUGPU, sizeof(double), hipMemcpyHostToDevice);
		sum02GPU01 << <dimGrid, dimBlock >> > (arrayGPU, numpiGPUCPU);
		hipDeviceSynchronize();
		dimBlock = { 1024 , 1 , 1 };
		timer = clock() - timer;
		hipMemcpy(numpiCPUGPU, numpiGPUCPU, sizeof(double), hipMemcpyDeviceToHost);
		printf(" - The value of PI in GPU is: %.8lf\n", numpiCPUGPU[0]); /*MAX DEC: 51*/
		printf(" - Total GPU time: %f ms.\n", ((((double)timer) / CLOCKS_PER_SEC) * 1000.0));
		printStats(timer, dimGrid, dimBlock);
		printf("==================================================\n");
		hipFree(arrayGPU);
		break;
	case 2: // Case02 - x Blocks with 1 Thread each.
		printf("==================================================\n");
		printf("\t\t<<<<< GPU >>>>>\n");
		dimGrid = { 1024 , 1 , 1 };
		dimBlock = { 1 , 1 , 1 };
		timer = clock();
		piGPU02 << <dimGrid, dimBlock >> > ();
		timer = clock() - timer;
		printStats(timer, dimGrid, dimBlock);
		printf("==================================================\n");
		break;
	case 3: // Case03 - x Blocks with m Threads each.
		printf("==================================================\n");
		printf("\t\t<<<<< GPU >>>>>\n");
		dimGrid = { 1024 , 1 , 1 };
		dimBlock = { 1024 , 1 , 1 };
		timer = clock();
		piGPU03 << <dimGrid, dimBlock >> > ();
		timer = clock() - timer;
		printStats(timer, dimGrid, dimBlock);
		printf("==================================================\n");
		break;
	case 4: // Case04 - x * y Blocks with 1 Thread each.
		printf("==================================================\n");
		printf("\t\t<<<<< GPU >>>>>\n");
		dimGrid = { 1024 , 1024 , 1 };
		dimBlock = { 1 , 1 , 1 };
		timer = clock();
		piGPU04 << <dimGrid, dimBlock >> > ();
		timer = clock() - timer;
		printStats(timer, dimGrid, dimBlock);
		printf("==================================================\n");
		break;
	case 5: // Case05 - 1 Block with m * n Threads each.
		printf("==================================================\n");
		printf("\t\t<<<<< GPU >>>>>\n");
		dimGrid = { 1 , 1 , 1 };
		dimBlock = { 30 , 30 , 1 };
		timer = clock();
		piGPU05 << <dimGrid, dimBlock >> > ();
		timer = clock() - timer;
		printStats(timer, dimGrid, dimBlock);
		printf("==================================================\n");
		break;
	case 6: // Case06 - x * y Blocks with m * n Threads each.
		printf("==================================================\n");
		printf("\t\t<<<<< GPU >>>>>\n");
		dimGrid = { 1024 , 1024 , 1 };
		dimBlock = { 30 , 30 , 1 };
		timer = clock();
		piGPU06 << <dimGrid, dimBlock >> > ();
		timer = clock() - timer;
		printStats(timer, dimGrid, dimBlock);
		printf("==================================================\n");
		break;
	default:
		printf(">>> INVALID OPTION <<<\n");
		return 0;
		break;
	}

	double numpiCPU = piCPU();

	system("pause");
	return 0;
}
