//////////////////////////////////////////////////
//          Project 01 PI Calculation           //
//////////////////////////////////////////////////
// C++
#include <cstdlib>
#include <cstdio>
#include <ctime>
#include <cmath>
// CUDA C / C++

#include <hip/hip_runtime.h>


const double iterations = 90000000; /*MAX: 94906264*/

__host__ double piCPU()
{
	double sum = 0;
	for (double i = 1; i < iterations; ++i)
	{
		sum += (1 / (i * i));
	}
	return sqrt(sum * 6);
}

int main(int argc, char* argv[])
{
	printf("==================================================\n");
	printf(" Calculation of PI on CPU and GPU.\n");
	printf(" > Number of iterations: %.1lf\n", iterations);
	printf("==================================================\n");
	printf("\t\t<<<CPU>>>\n");
	clock_t timer1 = clock();
	printf("The value of PI in CPU is: %.8lf\n", piCPU()); /*MAX DEC: 51*/
	timer1 = clock() - timer1;
	printf("Total CPU time: %f ms.\n", ((((double)timer1) / CLOCKS_PER_SEC) * 1000));
	printf("==================================================\n");

	system("pause");
	return 0;
}
