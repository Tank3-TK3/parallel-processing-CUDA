//////////////////////////////////////////////////
//          Project 01 PI Calculation           //
//////////////////////////////////////////////////
// C++
#include <cstdlib>
#include <cstdio>
#include <ctime>
#include <cmath>
// CUDA C / C++

#include <hip/hip_runtime.h>


const double iterations = 90000000; /*MAX: 94906264*/

__host__ int printDevProp()
{
	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, 0);
	printf("==================================================\n");
	printf(" >>>>>>> PI Calculation with CPU and GPU <<<<<<<\n");
	printf(" - Device Name: %s\n", devProp.name);
	printf(" - Maximum number of threads per block: %d\n", devProp.maxThreadsPerBlock);
	printf(" - Number of iterations: %.1lf\n", iterations);
	printf("==================================================\n");
	return devProp.maxThreadsPerBlock;
}

__host__ void piCPU()
{
	printf("\t\t<<<<< CPU >>>>>\n");
	double sum = 0;
	clock_t timer1 = clock();
	for (double i = 1; i < iterations; ++i)
	{
		sum += (1 / (i * i));
	}
	timer1 = clock() - timer1;
	printf(" - The value of PI in CPU is: %.8lf\n", sqrt(sum * 6)); /*MAX DEC: 51*/
	printf(" - Total CPU time: %f ms.\n", ((((double)timer1) / CLOCKS_PER_SEC) * 1000));
	printf("==================================================\n");
}

int main(int argc, char* argv[])
{
	hipSetDevice(0);

	int maxHilos = printDevProp();
	piCPU();

	system("pause");
	return 0;
}
