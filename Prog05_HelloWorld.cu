//////////////////////////////////////////////////
//           Program 05 Hello World             //
//////////////////////////////////////////////////
#include <cstdio>
#include <ctime>
#include <hip/hip_runtime.h>


__device__ void printInfo( int tid )
{
	printf("\t< I'm the thread (%d , %d , %d) of the block (%d , %d , %d) # %d\n" ,
	threadIdx.x , threadIdx.y , threadIdx.z , blockIdx.x , blockIdx.y , blockIdx.z , tid );
}

__global__ void case01()
{
	int tid = threadIdx.x;
	printInfo( tid );
}

__global__ void case02()
{
	int tid = blockIdx.x;
	printInfo( tid );
}

__global__ void case03()
{
	int tid = ( blockIdx.x * blockDim.x ) + threadIdx.x;
	printInfo( tid );
}

__global__ void case04()
{
	int tid = ( blockIdx.x * gridDim.y ) + blockIdx.y;
	printInfo( tid );
}

__global__ void case05()
{
	int tid = ( threadIdx.x * blockDim.y ) + threadIdx.y;
	printInfo( tid );
}

__global__ void case06()
{
	int numHilo = ( threadIdx.x * blockDim.y ) + threadIdx.y;
	int tid = ( blockIdx.x * blockDim.x * blockDim.y ) + numHilo;
	printInfo( tid );
}

__global__ void case07()
{
	int numBloque = ( blockIdx.x * gridDim.y ) + blockIdx.y;
	int tid = ( numBloque * blockDim.x ) + threadIdx.x;
	printInfo( tid );
}

__global__ void case08()
{
	int numBloque = ( blockIdx.x * gridDim.y ) + blockIdx.y;
	int numHilo = ( threadIdx.x * blockDim.y ) + threadIdx.y;
	int tid = ( numBloque * blockDim.x * blockDim.y ) + numHilo;
	printInfo( tid );
}

__global__ void case09()
{
	int numBloque = ( blockIdx.x * gridDim.y * gridDim.z ) + ( blockIdx.y * gridDim.z ) + blockIdx.z;
	int tid = numBloque + threadIdx.x;
	printInfo( tid );
}

__global__ void case10()
{
	int numBloque = ( blockIdx.x * gridDim.y * gridDim.z )  + ( blockIdx.y * gridDim.z ) + blockIdx.z;
	int tid = ( numBloque * blockDim.x ) + threadIdx.x;
	printInfo( tid );
}

__global__ void case11()
{
	int numBloque = ( blockIdx.x * gridDim.y * gridDim.z ) + ( blockIdx.y * gridDim.z ) + blockIdx.z;
	int numHilo = ( threadIdx.x * blockDim.y ) + threadIdx.y;
	int tid = ( numBloque * blockDim.x * blockDim.y ) + numHilo;
	printInfo( tid );
}

__global__ void case12()
{
	int numBloque = ( blockIdx.x * gridDim.y * gridDim.z ) + ( blockIdx.y * gridDim.z ) + blockIdx.z;
	int numHilo = ( threadIdx.x * blockDim.y * blockDim.z ) + ( threadIdx.y * blockDim.z ) + threadIdx.z;
	int tid = ( numBloque * blockDim.x * blockDim.y * blockDim.z ) + numHilo;
	printInfo( tid );
}

__host__ void printStats( clock_t timer , dim3 dimGrid , dim3 dimBlock )
{
	printf( "> The operation on the device took: %.3f ms.\n\n" , 
		( ( ( float ) timer ) / CLOCKS_PER_SEC ) * 1000 );
	hipDeviceSynchronize(); // Synchronize the GPU preventing premature termination
	printf( "\n- Total Threads: %d\n" , 
		dimGrid.x * dimGrid.y * dimGrid.z * dimBlock.x * dimBlock.y * dimBlock.z );
	printf( "- Configuracion de ejecucion: \n");
	printf( "\t+ Grid [%d, %d, %d] Bloque [%d, %d, %d]\n", 
		dimGrid.x, dimGrid.y, dimGrid.z, dimBlock.x, dimBlock.y, dimBlock.z);
}

__host__ int printMenuOpt()
{
	int opt = ' ';
	printf( "##################################################\n" );
	printf( "+ Write the number of the case you want to run:\n" );
	printf( "\t1) Case01 - 1 Block with 1 Thread.\n" );
	printf( "\t2) Case02 - n Blocks with 1 Thread each.\n" );
	printf( "\t3) Case03 - n Blocks with m Threads each.\n" );
	printf( "\t4) Case04 - n * m Blocks with 1 Thread each.\n" );
	printf( "\t5) Case05 - 1 Block with n * m Threads each.\n" );
	printf( "\t6) Case06 - n Blocks with m * r Threads each.\n" );
	printf( "\t7) Case07 - n * m Blocks with r Threads each.\n" );
	printf( "\t8) Case08 - n * m Blocks with r * s Threads each.\n" );
	printf( "\t9) Case09 - n * m * r Blocks with 1 Thread each.\n" );
	printf( "\t10) Case10 - n * m * r Blocks with p Threads each.\n" );
	printf( "\t11) Case11 - n * m * r Blocks with p * s Threads each.\n" );
	printf( "\t12) Case 12 - n * m * r Blocks with p * s * t Threads each.\n" );
	printf("> ");
	scanf( "%d" , &opt );
	return opt;
}

int main( int argc , char* argv[] )
{
	clock_t timer;
	dim3 dimGrid;
	dim3 dimBlock;

	// set the ID of the CUDA device
	hipSetDevice( 0 );
	
	switch ( printMenuOpt() )
	{
		case 1: // Case01 - 1 Block with 1 Thread.
				printf( "##################################################\n" );
				dimGrid = { 1 , 1 , 1 };
				dimBlock = { 1 , 1 , 1 };
				timer = clock();
				case01 <<< dimGrid , dimBlock >>>();
				timer = clock() - timer;
				printStats( timer , dimGrid , dimBlock );
				printf( "##################################################\n" );
			break;
		case 2: // Case02 - n Blocks with 1 Thread each.
				printf( "##################################################\n" );
				dimGrid = { 20 , 1 , 1 };
				dimBlock = { 1 , 1 , 1 };
				timer = clock();
				case02 <<< dimGrid , dimBlock >>>();
				timer = clock() - timer;
				printStats( timer , dimGrid , dimBlock );
				printf( "##################################################\n" );
			break;
		case 3: // Case03 - n Blocks with m Threads each.
				printf( "##################################################\n" );
				dimGrid = { 5 , 1 , 1 };
				dimBlock = { 4 , 1 , 1 };
				timer = clock();
				case03 <<< dimGrid , dimBlock >>>();
				timer = clock() - timer;
				printStats( timer , dimGrid , dimBlock );
				printf( "##################################################\n" );
			break;
		case 4: // Case04 - n * m Blocks with 1 Thread each.
				printf( "##################################################\n" );
				dimGrid = { 4 , 5 , 1 };
				dimBlock = { 1 , 1 , 1 };
				timer = clock();
				case04 <<< dimGrid , dimBlock >>>();
				timer = clock() - timer;
				printStats( timer , dimGrid , dimBlock );
				printf( "##################################################\n" );
			break;
		case 5: // Case05 - 1 Block with n * m Threads each.
				printf( "##################################################\n" );
				dimGrid = { 1 , 1 , 1 };
				dimBlock = { 4 , 5 , 1 };
				timer = clock();
				case05 <<< dimGrid , dimBlock >>>();
				timer = clock() - timer;
				printStats( timer , dimGrid , dimBlock );
				printf( "##################################################\n" );
			break;
		case 6: // Case06 - n Blocks with m * r Threads each.
				printf( "##################################################\n" );
				dimGrid = { 5 , 1 , 1 };
				dimBlock = { 3 , 2 , 1 };
				timer = clock();
				case06 <<< dimGrid , dimBlock >>>();
				timer = clock() - timer;
				printStats( timer , dimGrid , dimBlock );
				printf( "##################################################\n" );
			break;
		case 7: // Case07 - n * m Blocks with r Threads each.
				printf( "##################################################\n" );
				dimGrid = { 3 , 2 , 1 };
				dimBlock = { 4 , 1 , 1 };
				timer = clock();
				case07 <<< dimGrid , dimBlock >>>();
				timer = clock() - timer;
				printStats( timer , dimGrid , dimBlock );
				printf( "##################################################\n" );
			break;
		case 8: // Case08 - n * m Blocks with r * s Threads each.
				printf( "##################################################\n" );
				dimGrid = { 3 , 3 , 1 };
				dimBlock = { 2 , 2 , 1 };
				timer = clock();
				case08 <<< dimGrid , dimBlock >>>();
				timer = clock() - timer;
				printStats( timer , dimGrid , dimBlock );
				printf( "##################################################\n" );
			break;
		case 9: // Case09 - n * m * r Blocks with 1 Thread each.
				printf( "##################################################\n" );
				dimGrid = { 2 , 3 , 4 };
				dimBlock = { 1 , 1 , 1 };
				timer = clock();
				case09 <<< dimGrid , dimBlock >>>();
				timer = clock() - timer;
				printStats( timer , dimGrid , dimBlock );
				printf( "##################################################\n" );
			break;
		case 10: // Case10 - n * m * r Blocks with p Threads each.
				printf( "##################################################\n" );
				dimGrid = { 2 , 3 , 4 };
				dimBlock = { 3 , 1 , 1 };
				timer = clock();
				case10 <<< dimGrid , dimBlock >>>();
				timer = clock() - timer;
				printStats( timer , dimGrid , dimBlock );
				printf( "##################################################\n" );
			break;
		case 11: // Case11 - n * m * r Blocks with p * s Threads each.
				printf( "##################################################\n" );
				dimGrid = { 2 , 3 , 4 };
				dimBlock = { 2 , 3 , 1 };
				timer = clock();
				case11 <<< dimGrid , dimBlock >>>();
				timer = clock() - timer;
				printStats( timer , dimGrid , dimBlock );
				printf( "##################################################\n" );
			break;
		case 12: // Case12 - n * m * r Blocks with p * s * t Threads each.
				printf( "##################################################\n" );
				dimGrid = { 2 , 3 , 4 };
				dimBlock = { 2 , 2 , 3 };
				timer = clock();
				case12 <<< dimGrid , dimBlock >>>();
				timer = clock() - timer;
				printStats( timer , dimGrid , dimBlock );
				printf( "##################################################\n" );
			break;
		default:
				printf(">>> INVALID OPTION <<<\n");
				return 0;
			break;
	}
	
	system( "pause" );
	return 0;
}
