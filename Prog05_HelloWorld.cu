//////////////////////////////////////////////////
//           Program 05 Hello World             //
//////////////////////////////////////////////////
#include <cstdio>
#include <ctime>
#include <hip/hip_runtime.h>


__device__ void printInfo( int tid )
{
	printf("\t< I'm the thread (%d , %d , %d) of the block (%d , %d , %d) # %d\n" ,
	threadIdx.x , threadIdx.y , threadIdx.z , blockIdx.x , blockIdx.y , blockIdx.z , tid );
}

__global__ void case01()
{
	int tid = threadIdx.x;
	printInfo( tid );
}

__global__ void case02()
{
	int tid = blockIdx.x;
	printInfo( tid );
}

__global__ void case03()
{
	int tid = ( blockIdx.x * blockDim.x ) + threadIdx.x;
	printInfo( tid );
}

__global__ void case04()
{
	int tid = ( blockIdx.x * gridDim.y ) + blockIdx.y;
	printInfo( tid );
}

__global__ void case05()
{
	int tid = ( threadIdx.x * blockDim.y ) + threadIdx.y;
	printInfo( tid );
}

__global__ void case06()
{
	int numHilo = ( threadIdx.x * blockDim.y ) + threadIdx.y;
	int tid = ( blockIdx.x * blockDim.x * blockDim.y ) + numHilo;
	printInfo( tid );
}

__global__ void case07()
{
	int numBloque = ( blockIdx.x * gridDim.y ) + blockIdx.y;
	int tid = ( numBloque * blockDim.x ) + threadIdx.x;
	printInfo( tid );
}

__global__ void case08()
{
	int numBloque = ( blockIdx.x * gridDim.y ) + blockIdx.y;
	int numHilo = ( threadIdx.x * blockDim.y ) + threadIdx.y;
	int tid = ( numBloque * blockDim.x * blockDim.y ) + numHilo;
	printInfo( tid );
}

__global__ void case09()
{
	int numBloque = ( blockIdx.x * gridDim.y * gridDim.z ) + ( blockIdx.y * gridDim.z ) + blockIdx.z;
	int tid = numBloque + threadIdx.x;
	printInfo( tid );
}

__global__ void case10()
{
	int numBloque = ( blockIdx.x * gridDim.y * gridDim.z )  + ( blockIdx.y * gridDim.z ) + blockIdx.z;
	int tid = ( numBloque * blockDim.x ) + threadIdx.x;
	printInfo( tid );
}

__global__ void case11()
{
	int numBloque = ( blockIdx.x * gridDim.y * gridDim.z ) + ( blockIdx.y * gridDim.z ) + blockIdx.z;
	int numHilo = ( threadIdx.x * blockDim.y ) + threadIdx.y;
	int tid = ( numBloque * blockDim.x * blockDim.y ) + numHilo;
	printInfo( tid );
}

__global__ void case12()
{
	int numBloque = ( blockIdx.x * gridDim.y * gridDim.z ) + ( blockIdx.y * gridDim.z ) + blockIdx.z;
	int numHilo = ( threadIdx.x * blockDim.y * blockDim.z ) + ( threadIdx.y * blockDim.z ) + threadIdx.z;
	int tid = ( numBloque * blockDim.x * blockDim.y * blockDim.z ) + numHilo;
	printInfo( tid );
}

__host__ int printMenuOpt()
{
	int opt = ' ';
	printf( "##################################################\n" );
	printf( "+ Write the number of the case you want to run:\n" );
	printf( "\t1) Case01 - 1 Block with 1 Thread.\n" );
	printf( "\t2) Case02 - n Blocks with 1 Thread each.\n" );
	printf( "\t3) Case03 - n Blocks with m Threads each.\n" );
	printf( "\t4) Case04 - n * m Blocks with 1 Thread each.\n" );
	printf( "\t5) Case05 - 1 Block with n * m Threads each.\n" );
	printf( "\t6) Case06 - n Blocks with m * r Threads each.\n" );
	printf( "\t7) Case07 - n * m Blocks with r Threads each.\n" );
	printf( "\t8) Case08 - n * m Blocks with r * s Threads each.\n" );
	printf( "\t9) Case09 - n * m * r Blocks with 1 Thread each.\n" );
	printf( "\t10) Case10 - n * m * r Blocks with p Threads each.\n" );
	printf( "\t11) Case11 - n * m * r Blocks with p * s Threads each.\n" );
	printf( "\t12) Case 12 - n * m * r Blocks with p * s * t Threads each.\n" );
	printf("> ");
	scanf( "%d" , &opt );
	return opt;
}

int main( int argc , char* argv[] )
{
	clock_t timer;
	dim3 dimGrid;
	dim3 dimBlock;

	// set the ID of the CUDA device
	hipSetDevice( 0 );

	switch ( printMenuOpt() )
	{
		case 1: // Case01 - 1 Block with 1 Thread.
				printf( "##################################################\n" );
				dimGrid = { 1 , 1 , 1 };
				dimBlock = { 1 , 1 , 1 };
				timer = clock();
				case01 <<< dimGrid , dimBlock >>>();
				timer = clock() - timer;
				printf( "> The operation on the device took: %.3f ms.\n\n" , ( ( ( float ) timer ) / CLOCKS_PER_SEC ) * 1000 );
				hipDeviceSynchronize(); // Synchronize the GPU preventing premature termination
				printf( "\n- Total Threads: %d\n" , dimGrid.x * dimGrid.y * dimGrid.z * dimBlock.x * dimBlock.y * dimBlock.z );
				printf( "- Configuracion de ejecucion: \n");
				printf( "\t+ Grid [%d, %d, %d] Bloque [%d, %d, %d]\n", dimGrid.x, dimGrid.y, dimGrid.z, dimBlock.x, dimBlock.y, dimBlock.z);
				printf( "##################################################\n" );
			break;
		case 2: // Case02 - n Blocks with 1 Thread each.
			break;
		case 3: // Case03 - n Blocks with m Threads each.
			break;
		case 4:
			break;
		case 5:
			break;
		case 6:
			break;
		case 7:
			break;
		case 8:
			break;
		case 9:
			break;
		case 10:
			break;
		case 11:
			break;
		case 12:
			break;
		default:
				return 0;
			break;
	}

	// Caso 1 - 1 Bloque con 1 Hilo
	// dim3 dimGrid(1);
	// dim3 dimBlock(1);

	// Caso 2 - n Bloques con 1 Hilo c/u
	// dim3 dimGrid(20);
	// dim3 dimBlock(1);

	// Caso 3 - n Bloques con m Hilos c/u
	// dim3 dimGrid(5);
	// dim3 dimBlock(4);

	// Caso 4 - n x m Bloques con 1 Hilo c/u
	// dim3 dimGrid(4,5);
	// dim3 dimBlock(1);

	// Caso 5 - 1 Bloque con n x m Hilos c/u
	// dim3 dimGrid(1);
	// dim3 dimBlock(4,5);

	// Caso 6 - n Bloques con m x r Hilos c/u
	// dim3 dimGrid(5);
	// dim3 dimBlock(3, 2);

	// Caso 7 - n x m Bloques con r Hilos c/u
	// dim3 dimGrid(3, 2);
	// dim3 dimBlock(4);

	// Caso 8 - n x m Bloques con r x s Hilos c/u
	//dim3 dimGrid(3, 3);
	//dim3 dimBlock(2, 2);

	// Caso 9 - n x m x r Bloques con 1 Hilo c/u
	//dim3 dimGrid(2, 3, 4);
	//dim3 dimBlock(1);

	// Caso 10 - n x m x r Bloques con p Hilos c/u
	//dim3 dimGrid(2, 3, 4);
	//dim3 dimBlock(3);

	// Caso 11 - n x m x r Bloques con p x s Hilos c/u
	//dim3 dimGrid(2, 3, 4);
	//dim3 dimBlock(2, 3);

	// Caso 12 - n x m x r Bloques con p x s x t Hilos c/u
	// dim3 dimGrid(2, 3, 4);
	// dim3 dimBlock(2, 2, 3);
	
	system( "pause" );
	return 0;
}
