#include "hip/hip_runtime.h"
//////////////////////////////////////////////////
//           Program 05 Hello World             //
//////////////////////////////////////////////////
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>

using namespace std;

__global__ void hello_kernel()
{
	// Caso 1
	int tid = threadIdx.x;

	// Caso 2
	//int tid = blockIdx.x;

	// Caso 3
	//int tid = (blockIdx.x*blockDim.x)+threadIdx.x;

	// Caso 4
	//int numBloque = (blockIdx.x * gridDim.y) + blockIdx.y;
	//int tid = numBloque;

	// Caso 5
	//int numHilo = (threadIdx.x * blockDim.y) + threadIdx.y;
	//int tid = numHilo;

	// Caso 6
	//int numBloque = blockIdx.x;
	//int numHilo = (threadIdx.x * blockDim.y) + threadIdx.y;
	//int tid = (numBloque * blockDim.x * blockDim.y) + numHilo;

	// Caso 7
	//int numBloque = (blockIdx.x*gridDim.y)+blockIdx.y;
	//int numHilo =threadIdx.x;
	//int tid = (numBloque * blockDim.x) + numHilo;

	// Caso 8
	//int numBloque = (blockIdx.x*gridDim.y) + blockIdx.y;
	//int numHilo = (threadIdx.x * blockDim.y) + threadIdx.y;
	//int tid = (numBloque * blockDim.x * blockDim.y) + numHilo;

	// Caso 9
	//int numBloque = (blockIdx.x*gridDim.y*gridDim.z) + 
	//				(blockIdx.y*gridDim.z)+
	//				 blockIdx.z;
	//int numHilo = threadIdx.x;
	//int tid = numBloque + numHilo;

	// Caso 10
	//int numBloque = (blockIdx.x*gridDim.y*gridDim.z) +
	//	(blockIdx.y*gridDim.z) +
	//	blockIdx.z;
	//int numHilo = threadIdx.x;
	//int tid = (numBloque*blockDim.x) + numHilo;

	// Caso 11
	//int numBloque = (blockIdx.x*gridDim.y*gridDim.z) +
	//	(blockIdx.y*gridDim.z) +
	//	blockIdx.z;
	//int numHilo = (threadIdx.x * blockDim.y) + threadIdx.y;
	//int tid = (numBloque * blockDim.x * blockDim.y) 
	//	+ numHilo;

	// Caso 12
	// int numBloque = (blockIdx.x * gridDim.y * gridDim.z) +
	//     (blockIdx.y * gridDim.z) +
	//     blockIdx.z;
	// int numHilo = (threadIdx.x * blockDim.y * blockDim.z) +
	//     (threadIdx.y * blockDim.z) +
	//     threadIdx.z;
	// int tid = (numBloque * blockDim.x * blockDim.y * blockDim.z)
	//     + numHilo;

	// print a greeting message
	printf("Soy el hilo (%2d, %2d, %2d) del bloque (%2d, %2d, %2d) # %2d\n", threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z, tid);
	//printf("Hello from thread %d!\n", tid);
}

int main( int argc , char* argv[] )
{
	// Saludos desde el Host
	cout << "##################################################\n";
	cout << "\tHello, world from the host (CPU)!\n";
	cout << "##################################################\n";

	// set the ID of the CUDA device
	hipSetDevice( 0 );

	// Caso 1 - 1 Bloque con 1 Hilo
	dim3 dimGrid(1);
	dim3 dimBlock(1);

	// Caso 2 - n Bloques con 1 Hilo c/u
	//dim3 dimGrid(20);
	//dim3 dimBlock(1);

	// Caso 3 - n Bloques con m Hilos c/u
	//dim3 dimGrid(5);
	//dim3 dimBlock(4);

	// Caso 4 - n x m Bloques con 1 Hilo c/u
	//dim3 dimGrid(4,5);
	//dim3 dimBlock(1);

	// Caso 5 - 1 Bloque con n x m Hilos c/u
	//dim3 dimGrid(1);
	//dim3 dimBlock(4,5);

	// Caso 6 - n Bloques con m x r Hilos c/u
	//dim3 dimGrid(5);
	//dim3 dimBlock(3, 2);

	// Caso 7 - n x m Bloques con r Hilos c/u
	//dim3 dimGrid(3, 2);
	//dim3 dimBlock(4);

	// Caso 8 - n x m Bloques con r x s Hilos c/u
	//dim3 dimGrid(3, 3);
	//dim3 dimBlock(2, 2);

	// Caso 9 - n x m x r Bloques con 1 Hilo c/u
	//dim3 dimGrid(2, 3, 4);
	//dim3 dimBlock(1);

	// Caso 10 - n x m x r Bloques con p Hilos c/u
	//dim3 dimGrid(2, 3, 4);
	//dim3 dimBlock(3);

	// Caso 11 - n x m x r Bloques con p x s Hilos c/u
	//dim3 dimGrid(2, 3, 4);
	//dim3 dimBlock(2, 3);

	// Caso 12 - n x m x r Bloques con p x s x t Hilos c/u
	// dim3 dimGrid(2, 3, 4);
	// dim3 dimBlock(2, 2, 3);

	clock_t timer1 = clock();
	// invoke kernel using 4 threads executed in 1 thread block
	hello_kernel << < dimGrid, dimBlock >> > ();

	timer1 = clock() - timer1;
	printf("Operacion en Device toma %10.3f ms.\n", (((float)timer1) / CLOCKS_PER_SEC) * 1000);

	// synchronize the GPU preventing premature termination
	hipDeviceSynchronize();

	printf("\n");
	printf("Hilos totales: %d\n", dimGrid.x * dimGrid.y * dimGrid.z * dimBlock.x * dimBlock.y * dimBlock.z);

	printf("\n");
	printf("Configuracion de ejecucion: \n");
	printf("Grid [%d, %d, %d] Bloque [%d, %d, %d]\n", dimGrid.x, dimGrid.y, dimGrid.z, dimBlock.x, dimBlock.y, dimBlock.z);

	system( "pause" );
	return 0;
}
