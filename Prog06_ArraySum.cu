// suma de vectores
// Caso 1 length bloques con 1 hilo c/u  -- Maximo 63000
// Caso 2 1,length bloque con 1 hilo c/u -- Maximo 63000
// Restringido por numero maximo de hilos x bloque
// Caso 3 1 bloque con length hilos c/u  -- Maximo 1024
// Caso 4 length/1024 bloques con 1024 hilos c/u
// Caso 5 Maximizar el num de hilos por bloque cercano a 1024
// Caso 6 length/(1024 x ElemxHilo) bloques con 1024 hilos c/u
/////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////
//            Program 06 Array Sum              //
//////////////////////////////////////////////////
// C++
#include <cstdlib>
#include <cstdio>
#include <ctime>
// CUDA C / C++
#include <hip/hip_runtime.h>



#define length 100
#define epsilon float(0.0000001)
#define elemxHilo 5

__host__ int printDevProp()
{
	hipDeviceProp_t devProp;
	hipGetDeviceProperties( &devProp , 0 );
	printf( "##################################################\n" );
	printf( "- Device Name: %s\n" , devProp.name );
	printf( "- Maximum number of threads per block: %d\n" , devProp.maxThreadsPerBlock );
	printf( "##################################################\n" );
	return devProp.maxThreadsPerBlock;
}

__global__ void add( float *a , float *b , float *c )
{
	// Caso 1
	// int tid = blockIdx.x;  
	// c[tid] = a[tid] + b[tid];

	// Caso 2
	//int tid = blockIdx.y;
	//c[tid] = a[tid] + b[tid];

	// Caso 3
	//int tid = threadIdx.x;
	//c[tid] = a[tid] + b[tid];

	// Caso 4 y 5
	int tid = ( blockIdx.x * blockDim.x ) + threadIdx.x; 
	if ( tid < length ) c[tid] = a[tid] + b[tid];  

	// Caso 6
	//int tid = (blockIdx.x*blockDim.x*elemxHilo) +
	//	(threadIdx.x*elemxHilo);
	// int tid = ((blockIdx.x*blockDim.x) + threadIdx.x) *
	// 	elemxHilo;
	// for (int i = 0; i < elemxHilo; i++) {
	// 	if ((tid + i) < length)
	// 		c[tid + i] = a[tid + i] + b[tid + i];
	// }
}

float comparar( float *var1 , float *var2 , int *numDifer ) {
	float diferencia = 0;
	float difActual = 0;
	int cont = 0;
	for (int i = 0; i < length; i++) {
		difActual = abs(var1[i] - var2[i]);
		diferencia += difActual;
		if (difActual>epsilon)
			cont++;
	}
	*numDifer = cont;
	return diferencia;
}

void imprimir(float *var1, float *var2,
	float *result1, float *result2) {
	//display results
	for (int i = 0; i<length; i++) {
		printf("%5d. %10.3f + %10.3f = CPU %10.3f   GPU %10.3f\n",
			i + 1, var1[i], var2[i], result1[i], result2[i]);
	}
}

int divEntera(int n, int m) {
	int valor = 0;
	if ((n%m) == 0)
		valor = n / m;
	else
		valor = (n / m) + 1;
	return valor;
}

void addCPU( float *a , float *b , float *c )
{
	for( int i = 0 ; i < length ; ++i )
	{
		c[i] = a[i] + b[i];
	}
}

int main( int argc , char* argv[] )
{
	float a[length] , b[length] , gpu_c[length];
	float cpu_c[length];
	float *dev_a , *dev_b , *dev_c;
	float diferencia = 0;
	int nDiferentes = 0;

	memset( gpu_c , 0 , length * sizeof( float ) );
	memset( cpu_c , 0 , length * sizeof( float ) );

	hipSetDevice( 0 );

	int maxHilos = printDevProp(); // Get device properties

	// Allocate memory on device
	hipMalloc( ( void** ) &dev_a, length * sizeof( float ) );
	hipMalloc( ( void** ) &dev_b, length * sizeof( float ) );
	hipMalloc( ( void** ) &dev_c, length * sizeof( float ) );
	hipMemset( dev_c , 0 , length * sizeof( float ) );

	srand( ( unsigned ) time( NULL ) ); // Dynamic random seed
	
	// Fill arrays a and b on the CPU
	for( int i = 0 ; i < length ; ++i )
	{
		a[i] = ( ( ( float ) rand() / ( float ) RAND_MAX ) * 100 ) - 20;
		b[i] = ( ( ( float ) rand() / ( float ) RAND_MAX ) * 100 ) - 20;
	}

	printf( "- Array addition with %d elements.\n" , length );

	clock_t timer1 = clock();
	addCPU( a , b , cpu_c );
	timer1 = clock() - timer1;

	printf( "- Operations on the CPU takes %.3f ms.\n" ,
		( ( ( float ) timer1 ) / CLOCKS_PER_SEC ) * 1000 );
	// Copy memory from host to device
	// Destination , Source , Data Size , Copy Direction
	hipMemcpy( dev_a , a , length * sizeof( float ) , hipMemcpyHostToDevice );
	hipMemcpy( dev_b , b , length * sizeof( float ) , hipMemcpyHostToDevice );

	clock_t timer2 = clock();
	// Caso 1
	// dim3 dimGrid( length );
	// dim3 dimBlock( 1 );

	// Caso 2
	//dim3 dimGrid( 1 , length );
	//dim3 dimBlock( 1 );

	//Caso 3 
	//dim3 dimGrid( 1 );
	//dim3 dimBlock( length );

	//Caso 4
	dim3 dimGrid(divEntera( length , maxHilos ) );
	dim3 dimBlock( maxHilos );

	//Caso 5
	//int numBloques = divEntera( length , maxHilos );
	//int numHilos = divEntera( length , numBloques );
	//dim3 dimGrid( numBloques );
	//dim3 dimBlock( numHilos );

	//Caso 6
	// int numBloques = divEntera( length , maxHilos * elemxHilo );
	// dim3 dimGrid( numBloques );
	// dim3 dimBlock( maxHilos );

	hipError_t cudaStatus;
	add <<< dimGrid , dimBlock >>> ( dev_a , dev_b , dev_c );
	cudaStatus = hipGetLastError();
	if( cudaStatus != hipSuccess )
	{
		fprintf( stderr , "Kernel launch FAILED: %s\n" , hipGetErrorString( cudaStatus ) );
	}
	timer2 = clock() - timer2;

	printf( "- Operation on Device takes %.3f ms.\n" ,
		( ( ( float ) timer2 ) / CLOCKS_PER_SEC ) * 1000 );

	printf( "Configuracion de ejecucion: \n" );
	printf( "Grid [%d, %d, %d] Bloque [%d, %d, %d]\n" ,
		dimGrid.x , dimGrid.y , dimGrid.z , dimBlock.x , dimBlock.y , dimBlock.z );
	printf( "Elementos por hilo: %d\n" , elemxHilo );

	//copy results back - device to host
	// (destino, origen, tamaño datos, dirección de copiado)
	hipMemcpy(gpu_c, dev_c, length * sizeof(float),
		hipMemcpyDeviceToHost);

	imprimir(a, b, cpu_c, gpu_c);

	diferencia = comparar(gpu_c, cpu_c, &nDiferentes);

	printf( "Elementos diferentes %d (%.3f %%) Con valor de %.20f\n" ,
		nDiferentes , ( ( nDiferentes / float( length ) ) * 100 ) , diferencia );
	printf( "##################################################\n" );

	//free memory - GPU
	hipFree( dev_a );
	hipFree( dev_b );
	hipFree( dev_c );

	system( "pause" );
	return 0;
}
